/*
 ============================================================================
 Name        : CudaPasswordBruteForceCrack.cu
 Author      : Ettore
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>


#include "crypt.h"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}
__constant__ char dictionary[10]={'0','1','2','3','4','5','6','7','8','9'};
__global__ void kernel(char** results, char** hashes, int dim) {
	int mI = threadIdx.y+blockIdx.y*blockDim.y;
	int yI = threadIdx.x+blockIdx.x*blockDim.x + 1940;
	int dI = threadIdx.z;

	//conversion from int to char


	//days
	char dd[2];
	if(dI<10){
		dd[0]='0';
		dd[1]=dictionary[dI];
	}
	else{
		int tens = dI/10;
		dd[0]=dictionary[tens];
		int units = dI%10;
		dd[1]=dictionary[units];
	}
	//months
	char mm[2];
	if(mI<10){
			mm[0]='0';
			mm[1]=dictionary[mI];
		}
		else{
			int tens = mI/10;
			mm[0]=dictionary[tens];
			int units = mI%10;
			mm[1]=dictionary[units];
		}
	//years -suppose yI=1996
	char yyyy[4];
	int thousands = yI/1000; //yI/1000=1 poichè è int
	yyyy[0]=dictionary[thousands];
	int tmp = yI%1000; //yI%1000=996
	int hundreds = tmp/100; //996/100 = 9
	yyyy[1]=dictionary[hundreds];
	tmp = tmp%100; // 996%100=96
	int tens= tmp/10;//96/10=9
	yyyy[2]=dictionary[tens];
	tmp=tmp%10;//96%10 = 6
	int units = tmp;//6
	yyyy[3]=dictionary[units];

	//end conversion


	char yyyymmdd[9] = {yyyy[0],yyyy[1],yyyy[2],yyyy[3],mm[0],mm[1],dd[0],dd[1],0};
	printf("%s \n", yyyymmdd);
	for(int i=0; i<dim; i++){
   /* char* pwd="";
    char* psw;
    char* salt = "parallel";
    psw = crypt("Ettore", salt);*/
    /*if (hashes[i] == crypt(pwd,"parallel")){
      results[i]=pwd;
    }*/
  }

}


int main(void)
{
  #define dim 100
  char * resultsHost[dim];
  char ** hashes;
  char ** results;
  FILE * fp;
  char * line = NULL;
  size_t len = 0;
  ssize_t read;
  char* hashesHost[dim];
  int k=0;
  fp = fopen("PswDb/db100.txt", "r");
  while ((read = getline(&line, &len, fp)) != -1) {
    char* hash =(char*) malloc(sizeof(char)*13);
    for(int i = 0; i<13; i++){
      hash[i]=line[i+9];
    }
    hashesHost[k]=hash;
    k++;
  }
  fclose(fp);
  free(line);
  char* psw;
  char* salt = "parallel";
  //psw = crypt("19961024", salt);
  std::cout<< "19961024 " << psw << "\n";
  printf("--- %s\n", hashesHost[0]);

  // allocate device memory


  CUDA_CHECK_RETURN(
		  hipMalloc((void **) &hashes, sizeof(char) * 13 * dim));

  CUDA_CHECK_RETURN(
		  hipMalloc((void **) &results, sizeof(char) * 13 * dim));

  // copy from host to device memory
  CUDA_CHECK_RETURN(
      hipMemcpy(hashes, hashesHost, dim * 13 * sizeof(char),
          hipMemcpyHostToDevice));


  //@@ INSERT CODE HERE
  dim3 dimGrid(7,4);
  dim3 dimBlock(10,3,31);
  kernel<<<dimGrid, dimBlock>>>(results,hashes,dim);
  // copy results from device memory to host

  CUDA_CHECK_RETURN(
      hipMemcpy(resultsHost, results, dim * 13 * sizeof(char),
          hipMemcpyDeviceToHost));
  hipFree(hashes);
  hipFree(results);
  return 0;
}
