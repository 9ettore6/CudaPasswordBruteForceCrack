/*
 ============================================================================
 Name        : CudaPasswordBruteForceCrack.cu
 Author      : Ettore
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>


#include "crypt.h"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}

__global__ void kernel(char** results, char** hashes, int dim) {
  for(int i=0; i<dim; i++){
	int mI = threadIdx.y+blockIdx.y*blockDim.y;
	int yI = threadIdx.x+blockIdx.x*blockDim.x;
	int dI = threadIdx.z;
	char m = mI +'0';//da int a char
	char y = yI +'0';
	char d = dI +'0';
    char yyyy[12];
    char mm[12];
    char dd[12];
    char* pwd="";
    /*if (hashes[i] == crypt(pwd,"parallel")){
      results[i]=pwd;
    }*/
  }

}


int main(void)
{
  #define dim 100
  char * resultsHost[dim];
  char * hashes[dim];
  char * results[dim];
  FILE * fp;
  char * line = NULL;
  size_t len = 0;
  ssize_t read;
  char* hashesHost[dim];
  int k=0;
  fp = fopen("PswDb/db100.txt", "r");
  while ((read = getline(&line, &len, fp)) != -1) {
    char* hash =(char*) malloc(sizeof(char)*13);
    for(int i = 0; i<13; i++){
      hash[i]=line[i+9];
    }
    hashesHost[k]=hash;
    k++;
  }
  fclose(fp);
  free(line);
  char* psw;
  char* salt = "parallel";
  psw = crypt("Ettore", salt);
  std::cout<< "Ettore " << psw << "\n";
  printf("--- %s\n", hashesHost[0]);
  printf("--- %s\n", hashesHost[1]);
  printf("--- %s\n", hashesHost[11]);
  // allocate device memory
  CUDA_CHECK_RETURN(
		  hipMalloc((void **) &hashes, sizeof(char) * 13 * dim));

  CUDA_CHECK_RETURN(
		  hipMalloc((void **) &results, sizeof(char) * 13 * dim));

  // copy from host to device memory
  CUDA_CHECK_RETURN(
      hipMemcpy(hashesHost, hashes, dim * 13 * sizeof(char),
          hipMemcpyHostToDevice));


  //@@ INSERT CODE HERE
  dim3 dimGrid(7,4);
  dim3 dimBlock(10,3,31);
  kernel<<<dimGrid, dimBlock>>>(results,hashes,dim);
  // copy results from device memory to host

  CUDA_CHECK_RETURN(
      hipMemcpy(results, resultsHost, dim * 13 * sizeof(char),
          hipMemcpyDeviceToHost));
  hipFree(hashes);
  hipFree(results);
  return 0;
}
