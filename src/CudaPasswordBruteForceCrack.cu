#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CudaPasswordBruteForceCrack.cu
 Author      : Culo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>


#include "crypt.h"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}
__constant__ char dictionary[10]={'0','1','2','3','4','5','6','7','8','9'};
__global__ void kernel(char** hashes, int dim, char** _a) {
	int mI = threadIdx.y+blockIdx.y*blockDim.y;
	int yI = threadIdx.x+blockIdx.x*blockDim.x + 1940;
	int dI = threadIdx.z;

	//conversion from int to char


	//days
	char dd[2];
	if(dI<10){
		dd[0]='0';
		dd[1]=dictionary[dI];
	}
	else{
		int tens = dI/10;
		dd[0]=dictionary[tens];
		int units = dI%10;
		dd[1]=dictionary[units];
	}
	//months
	char mm[2];
	if(mI<10){
			mm[0]='0';
			mm[1]=dictionary[mI];
		}
		else{
			int tens = mI/10;
			mm[0]=dictionary[tens];
			int units = mI%10;
			mm[1]=dictionary[units];
		}
	//years -suppose yI=1996
	char yyyy[4];
	int thousands = yI/1000; //yI/1000=1 poichè è int
	yyyy[0]=dictionary[thousands];
	int tmp = yI%1000; //yI%1000=996
	int hundreds = tmp/100; //996/100 = 9
	yyyy[1]=dictionary[hundreds];
	tmp = tmp%100; // 996%100=96
	int tens= tmp/10;//96/10=9
	yyyy[2]=dictionary[tens];
	tmp=tmp%10;//96%10 = 6
	int units = tmp;//6
	yyyy[3]=dictionary[units];

	//end conversion


	char yyyymmdd[9] = {yyyy[0],yyyy[1],yyyy[2],yyyy[3],mm[0],mm[1],dd[0],dd[1],0};
	printf("%s \n",_a[70]);
	for(int i=0; i<dim; i++){
   /* char* pwd="";
    char* psw;
    char* salt = "parallel";
    psw = crypt("Ettore", salt);*/
    /*if (hashes[i] == crypt(pwd,"parallel")){
      results[i]=pwd;
    }*/
  }

}


int main(void)
{
	#define dim 100
	char * resultsHost[dim];
	char ** results;
	FILE * fp;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	char* hashesHost[dim];
	int k=0;
	fp = fopen("PswDb/db100.txt", "r");
	while ((read = getline(&line, &len, fp)) != -1) {
	char* hash =(char*) malloc(sizeof(char)*14);
	for(int i = 0; i<13; i++){
	  hash[i]=line[i+9];
	}
	hash[13]=0;
	hashesHost[k]=hash;
	k++;
	}
	fclose(fp);
	free(line);
	char* psw;
	char* salt = "parallel";
	//psw = crypt("19961024", salt);
	printf("--- %s\n", hashesHost[0]);


	char * _s[dim];
	char ** _a;

	for (int i = 0; i < 100; i++) {

		CUDA_CHECK_RETURN( hipMalloc((void **)&_s[i], 13 * sizeof(char)) );

		CUDA_CHECK_RETURN( hipMemcpy(_s[i], hashesHost[i], 13 * sizeof(char), hipMemcpyHostToDevice) );

	  }
	CUDA_CHECK_RETURN( hipMalloc((void ***)&_a, dim * sizeof(char*)) );

	CUDA_CHECK_RETURN( hipMemcpy(_a, _s, dim * sizeof(char*), hipMemcpyHostToDevice) );




	CUDA_CHECK_RETURN(
		  hipMalloc((void **) &results, sizeof(char) * 13 * dim));




	//@@ INSERT CODE HERE
	dim3 dimGrid(7,4);
	dim3 dimBlock(10,3,31);
	kernel<<<dimGrid, dimBlock>>>(results,dim,_a);
	printf("culo");
	// copy results from device memory to host

	CUDA_CHECK_RETURN(
	  hipMemcpy(resultsHost, results, dim * 13 * sizeof(char),
		  hipMemcpyDeviceToHost));

	hipFree(results);
	return 0;
}
