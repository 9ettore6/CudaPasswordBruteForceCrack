#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CudaPasswordBruteForceCrack.cu
 Author      : CelozziCiabini
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "crypt.h"
#include "c_utils.h"
#include "des.h"
#include "des_utils.h"
#include "bit_utils.h"
#include "des_consts.h"
#include "des_kernel.h"
#include "cuda_utils.h"


static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}
//__constant__ char dictionary[10]={'0','1','2','3','4','5','6','7','8','9'};
__global__ void kernel(int* resultsDevice, int dim, uint64_t* hashesDevice, uint64_t* testTh) {
	int mI = threadIdx.y+blockIdx.y*blockDim.y;
	int yI = threadIdx.x+blockIdx.x*blockDim.x + 1940;
	int dI = threadIdx.z;
	//resultsDevice[(yI-1940)*12*31+mI*31+dI] = 1;
	uint64_t key = yI*10000+mI*100+dI;
	uint64_t encoded = 0;
	encoded = full_des_encode_block(key, key);
	for(int i=0;i<dim;i++){
		//printf("%d -- %d\n", hashesDevice[i], encoded);
		if (hashesDevice[i] == encoded){
			resultsDevice[i] = 1;
		}else{
			testTh[(yI-1940)*12*31+mI*31+dI] = key;
		}
	}
}


int main(void)
{
	#define dim 100
	int resultsHost[dim];
	uint64_t hostTestTH[26040];
	FILE * fp;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	uint64_t hashesHost[dim];
	int k=0;
	fp = fopen("PswDb/db100.txt", "r");
	while ((read = getline(&line, &len, fp)) != -1) {
		char* hash =(char*) malloc(sizeof(char)*9);
		for(int i = 0; i<9; i++){
		  hash[i]=line[i];
		}
		hash[8]= '\0'; //string termination
		hashesHost[k]=full_des_encode_block(atoi(hash),atoi(hash));
		k++;
	}
	fclose(fp);
	free(line);

	//GPU memory allocation
	uint64_t* hashesDevice;
	int* resultsDevice;
	uint64_t* testTh;

	CUDA_CHECK_RETURN( hipMalloc((void **)&hashesDevice, dim * sizeof(uint64_t)) );

	CUDA_CHECK_RETURN( hipMemcpy(hashesDevice, hashesHost, dim * sizeof(uint64_t), hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc((void **) &resultsDevice, sizeof(int) * dim));

	CUDA_CHECK_RETURN( hipMalloc((void **) &testTh, sizeof(uint64_t) * 26040));
	//@@ INSERT CODE HERE
	dim3 dimGrid(7,4);
	dim3 dimBlock(10,3,31);//
	kernel<<<dimGrid,dimBlock>>>(resultsDevice,dim,hashesDevice,testTh);
	// copy results from device memory to host
	/*for(int i = 0; i < dim; i++){
		resultsHost[i] = 0;
	}*/
	//printf("***********%d\n",resultsHost[5]);
	CUDA_CHECK_RETURN(
	  hipMemcpy(resultsHost, resultsDevice, dim * sizeof(int),
		  hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(
		  hipMemcpy(hostTestTH, testTh, dim * sizeof(uint64_t),
			  hipMemcpyDeviceToHost));
	//printf("***********%d\n",resultsHost[15]);
	hipFree(hashesDevice);
	hipFree(resultsDevice);

	int count = 0;
	int countff = 0;
	for(int i = 0; i < dim; i++){
		if(resultsHost[i] == 1){
			count++;
			printf("hash ok: %d\n", resultsHost[i]);
		}else{
			countff++;
			printf("hash nope: %d\n", hostTestTH[i]);
		}
	}
	printf("ccc: %d\n", count);
	printf("fff: %d", countff);
	return 0;
}
