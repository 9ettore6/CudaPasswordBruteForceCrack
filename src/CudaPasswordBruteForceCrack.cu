#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CudaPasswordBruteForceCrack.cu
 Author      : CelozziCiabini
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "crypt.h"
#include "c_utils.h"
#include "des.h"
#include "des_utils.h"
#include "bit_utils.h"
#include "des_consts.h"
#include "des_kernel.h"
#include "cuda_utils.h"


static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}
//__constant__ char dictionary[10]={'0','1','2','3','4','5','6','7','8','9'};
__global__ void kernel(int* resultsDevice, int dim, u_int64_t* hashesDevice) {
	int mI = threadIdx.y+blockIdx.y*blockDim.y;
	int yI = threadIdx.x+blockIdx.x*blockDim.x + 1940;
	int dI = threadIdx.z;
	//printf("%d \n",(yI-1940)*12*31+mI*31+dI);
	//printf("---%d, ---%d, --  %d\n", yI,mI,dI);
	//resultsDevice[(yI-1940)*12*31+mI*31+dI] = 1;
	uint64_t block = yI*10000+mI*100+dI;
	uint64_t encoded = full_des_encode_block(block, block);
	//test
	for(int i=0;i<dim;i++){
		if (hashesDevice[i] == encoded){
			resultsDevice[i]=1;
		}else{
			printf("hash %d -- enc %d -- blk %d -- it: %d\n", hashesDevice[i], encoded, block, i);
		}
	}
	/*
	//days
	char dd[2];
	if(dI<10){
		dd[0]='0';
		dd[1]=dictionary[dI];
	}
	else{
		int tens = dI/10;
		dd[0]=dictionary[tens];
		int units = dI%10;
		dd[1]=dictionary[units];
	}
	//months
	char mm[2];
	if(mI<10){
			mm[0]='0';
			mm[1]=dictionary[mI];
		}
		else{
			int tens = mI/10;
			mm[0]=dictionary[tens];
			int units = mI%10;
			mm[1]=dictionary[units];
		}
	//years -suppose yI=1996
	char yyyy[4];
	int thousands = yI/1000; //yI/1000=1 poichè è int
	yyyy[0]=dictionary[thousands];
	int tmp = yI%1000; //yI%1000=996
	int hundreds = tmp/100; //996/100 = 9
	yyyy[1]=dictionary[hundreds];
	tmp = tmp%100; // 996%100=96
	int tens= tmp/10;//96/10=9
	yyyy[2]=dictionary[tens];
	tmp=tmp%10;//96%10 = 6
	int units = tmp;//6
	yyyy[3]=dictionary[units];

	//end conversion


	char yyyymmdd[9] = {yyyy[0],yyyy[1],yyyy[2],yyyy[3],mm[0],mm[1],dd[0],dd[1],0};*/
	//printf("%d device \n",hashesDevice[55]); //test hipMemcpy
	/*for(int i=0; i<dim; i++){
		uint64_t block = yI*10000+mI*100+dI; //0x0123456789ABCDEF;
		//printf("block %d", block);
		uint64_t encoded = full_des_encode_block(block, block);
		//printf("en: %d", encoded);

		if (hashesDevice[i] == encoded){
			printf("hash: %d  enc: %d ----YEP \n",hashesDevice[i], encoded);
			break;
	    //  resultsDevice[i]=yyyymmdd;
	    }
		else{
			printf("hash: %d  enc: %d ----NOPE \n",hashesDevice[i], encoded);
		}
	}*/
}


int main(void)
{
	#define dim 100
	int resultsHost[dim];
	FILE * fp;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	u_int64_t hashesHost[dim];
	int k=0;
	fp = fopen("PswDb/db100.txt", "r");
	while ((read = getline(&line, &len, fp)) != -1) {
		char* hash =(char*) malloc(sizeof(char)*9);
		for(int i = 0; i<9; i++){
		  hash[i]=line[i];
		}
		hash[8]= '\0'; //string termination
		hashesHost[k]=full_des_encode_block(atoi(hash),atoi(hash));
		k++;
	}
	fclose(fp);
	free(line);

	//GPU memory allocation
	u_int64_t* hashesDevice;
	int* resultsDevice;

	CUDA_CHECK_RETURN( hipMalloc((void **)&hashesDevice, dim * sizeof(u_int64_t)) );

	CUDA_CHECK_RETURN( hipMemcpy(hashesDevice, hashesHost, dim * sizeof(u_int64_t), hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN(
			  hipMalloc((void **) &resultsDevice, sizeof(int) * dim));

	//@@ INSERT CODE HERE
	dim3 dimGrid(7,4);
	dim3 dimBlock(10,3,31);
	kernel<<<dimGrid,dimBlock>>>(resultsDevice,dim,hashesDevice);
	// copy results from device memory to host

	CUDA_CHECK_RETURN(
	  hipMemcpy(resultsHost, resultsDevice, dim * sizeof(int),
		  hipMemcpyDeviceToHost));

	hipFree(hashesDevice);
	hipFree(resultsDevice);
	int count = 0;
	for(int i = 0; i < dim; i++){
		if(resultsHost[i]==1)
			count++;
	}
	printf("ccc: %d", count);
	return 0;
}
