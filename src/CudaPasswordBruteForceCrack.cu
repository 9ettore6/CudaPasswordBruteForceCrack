#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : CudaPasswordBruteForceCrack.cu
 Author      : CelozziCiabini
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "crypt.h"
#include "c_utils.h"
#include "des.h"
#include "des_utils.h"
#include "bit_utils.h"
#include "des_consts.h"
#include "des_kernel.h"
#include "cuda_utils.h"


static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}

__global__ void kernel(int* resultsDevice, int dim, uint64_t* hashesDevice) {

	int date = threadIdx.x+blockDim.x*blockIdx.x;
	int year=(date/417)+1940;
	int month=((date%417)/32);
	int day=(date%32);
	uint64_t key = year*10000+month*100+day;
	uint64_t encoded = 0;
	encoded = full_des_encode_block(key, key);
	if(date==29690)
		printf("data: %d \n", key);
	if(month == 0 || day == 0){
	}else{
		if(date==29120)
			printf("%d \n",key);
		for(int i=0;i<dim;i++){
			if (hashesDevice[i] == encoded){
				resultsDevice[i] = 1;
			}
		}
	}
}


int main(void)
{
	#define dim 500
	int resultsHost[dim];
	FILE * fp;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	uint64_t hashesHost[dim];
	int k=0;
	fp = fopen("PswDb/db500.txt", "r");
	while ((read = getline(&line, &len, fp)) != -1) {
		char* hash =(char*) malloc(sizeof(char)*9);
		for(int i = 0; i<9; i++){
		  hash[i]=line[i];
		}
		hash[8]= '\0'; //string termination
		hashesHost[k]=full_des_encode_block(atoi(hash),atoi(hash));
		k++;
	}
	fclose(fp);
	free(line);

	//GPU memory allocation
	uint64_t* hashesDevice;
	int* resultsDevice;

	CUDA_CHECK_RETURN( hipMalloc((void **)&hashesDevice, dim * sizeof(uint64_t)) );

	CUDA_CHECK_RETURN( hipMemcpy(hashesDevice, hashesHost, dim * sizeof(uint64_t), hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc((void **) &resultsDevice, sizeof(int) * dim));
	//My machine is currently running on 3SM & 128 cudaCore/SM
	//@@ INSERT CODE HERE

	clock_t start = clock();
	kernel<<<232,128>>>(resultsDevice,dim,hashesDevice);
	// copy results from device memory to host

	hipDeviceSynchronize();
	CUDA_CHECK_RETURN(
		  hipMemcpy(resultsHost, resultsDevice, dim * sizeof(int),
			  hipMemcpyDeviceToHost));
	clock_t end = clock();
	float seconds = (float) (end - start) / CLOCKS_PER_SEC;
	hipFree(hashesDevice);
	hipFree(resultsDevice);

	int count = 0;
	for(int i = 0; i < dim; i++){
		if(resultsHost[i] == 1){
			count++;
		}
	}
	printf("ccc: %d\n",count);
	printf("time: %f",seconds);
	return 0;
}
